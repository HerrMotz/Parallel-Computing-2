#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


//increment wird auf host und device definiert
__host__ __device__ int increment(int a) 
{
     return a+1;
}
__host__ __device__ int add(int a, int b)
{
	return a+b;
}

__global__ void kernel(int *a, int *b, int size)
{
    int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse
    
    if (i<size)               //Fehlerbehandlung
      b[i]=add(a[i], b[i]);   //Increment
}


//Fülle A mit Werten
void fillA(int *a, int size)
{
   for (int i=0;i<size;i++)
      a[i]=i;
}

//Überprüfe Ergebnisse 
bool checkResult(int *a, int *b, int *c, int size)
{
   bool res=true;
   printf("a b c\n");
   for (int i=0;i<size;i++)
   {
      res&=(add(a[i],b[i])==c[i]);
      if (i<10) printf("%i %i %i\n",a[i],b[i],c[i]);
   }
   if (res) printf("Test passed\n");
   else printf("Test failed\n");
   return res;
}

int main(int argc, char**argv)
{
   //Problemgröße
   int size=1024;
   //Pointer auf Host/Device Speicher
   int *a_host, *b_host, *c_host, *a_dev, *b_dev;

   //Allokiere Host-Speicher
   a_host = (int*)malloc(size*sizeof(int));
   b_host = (int*)malloc(size*sizeof(int));
   c_host = (int*)malloc(size*sizeof(int));
   fillA(a_host,size);
   fillA(b_host,size);
   
   //Allokiere Device Speicher
   //Achtung: (void**)& sehr wichtig
   hipMalloc((void**)&a_dev,size*sizeof(int));
   hipMalloc((void**)&b_dev,size*sizeof(int));

   //Kopiere Host->Device
   hipMemcpy(a_dev,a_host,size*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(b_dev,b_host,size*sizeof(int),hipMemcpyHostToDevice);

   //Konfguration des Kernels (nur 1 Dimensional): 
   //256 Threads pro Threadblock
   //Mehrere Dimensionen möglich über dim3 threads(x,y,z); -> threads.x,threads.y,threads.z
   dim3 threads(256);
   //1024/256 = 4 Threadblöcke
   //Mehrere Dimensionen wie bei Threads möglich
   dim3 grid(size/threads.x);

   //Starte Kernel mit Konfiguration <<<grid,threads>>> auf Device Speicher
   //Wichtig: Spitze Klammern <<<>>> nicht vergessen!
   //Kernel wird asynchron zu CPU ausgeführt, d.h. hier könnte die CPU noch Arbeit verrichten
   kernel<<<grid,threads>>>(a_dev,b_dev,size);

   //Kopiere Ergebnis zurück (implizite Synchronisierung)
   hipMemcpy(c_host,b_dev,size*sizeof(int),hipMemcpyDeviceToHost);
   checkResult(a_host,b_host,c_host,size);

   //Gib Speicher wieder frei
   hipFree(a_dev);
   hipFree(b_dev);
   free(a_host);
   free(b_host);
   free(c_host);
   return 0;
}
